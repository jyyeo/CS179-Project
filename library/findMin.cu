#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__device__ static float atomicMin(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMax(float *dev_arr, int size, float *dev_min_val) {
	extern __shared__ float shmem[];
	const unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[i];

		__syncthreads();

		for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] > shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMax(dev_min_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMin(float *dev_arr, int size, float *dev_min_val) {
	
	findMax<<<1, size, size * sizeof(float)>>>(dev_arr, size, dev_min_val);

}
