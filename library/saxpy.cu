#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "saxpy.cuh"

__global__ 
void saxpy (float a, float *x, float *y, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		y[i] = a * x[i] + y[i];
	}
}

void cudaSaxpy (float a, float *x, float *y, int size) {
	float *dev_x;
	float *dev_y;

	hipMalloc((void**)&dev_x, size * sizeof(float));
	hipMalloc((void**)&dev_y, size * sizeof(float));
	
	hipMemcpy(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<1, size>>>(a, dev_x, dev_y, size);

	hipMemcpy(y, dev_y, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_x);
	hipFree(dev_y);
}