#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMin.cuh"

// __device__ static float atomicMin(float* address, float val) {
//     int* address_as_i = (int*) address;
//     int old = *address_as_i, assumed;
//     do {
//         assumed = old;
//         old = ::atomicCAS(address_as_i, assumed,
//             __float_as_int(::fminf(val, __int_as_float(assumed))));
//     } while (assumed != old);
//     return __int_as_float(old);
// }

__device__ __forceinline__ float atomicMin (float * addr, float value) {
        float old;
        old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
             __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

        return old;
}

__global__ void findMin(float *dev_arr, int size, float *dev_min_val) {
	extern __shared__ float shmem[];
	const unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size + 1) {
		shmem[tid] = dev_arr[i];

		__syncthreads();

		for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] > shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMin(dev_min_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
	if (tid == 0) {
		atomicMin(dev_min_val,shmem[0]);
	}

}

void cudaFindMin(float *arr, int size, float *min_val) {
	float *dev_arr;
	float *dev_min_val;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_min_val, sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dev_min_val, 0.0, sizeof(float));

	float block_dim = powf(2, ceil(log2(size)));
	
	findMin<<<1, block_dim, block_dim * sizeof(float)>>>(dev_arr, size, dev_min_val);
	
	hipMemcpy(min_val, dev_min_val, sizeof(float), hipMemcpyDeviceToHost);	
	hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_arr);
	hipFree(dev_min_val);

}
