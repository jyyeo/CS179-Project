#include "hip/hip_runtime.h"
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "findMax.cuh"

__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__
void
cudaMaximumKernel(hipfftComplex *out_data, float *max_abs_val,
    int padded_length) {

    extern __shared__ float shared_memory[];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.x;
    while (i < padded_length) {
	shared_memory[j] = fabs(out_data[j].x);
	__syncthreads();
	for (int k = blockDim.x; k > 1; k >>= 1) {
	    if (j < k) { //place values in contiguous memory locations
		if (shared_memory[j] < shared_memory[j + k]) { //store the larger value
		    shared_memory[j] = shared_memory[j + k];
		}
	    }		
	    __syncthreads();
	}
	if (threadIdx.x == 0) {
		atomicMax(max_abs_val, shared_memory[0]);
	} //run this once, after the largest value is in first memory location
    	i += blockDim.x * gridDim.x;
    }
}

void cudaCallMaximumKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        hipfftComplex *out_data,
        float *max_abs_val,
        const unsigned int padded_length) {
        

    /* TODO 2: Call the max-finding kernel. */
    cudaMaximumKernel<<<blocks,threadsPerBlock>>>(out_data, max_abs_val, padded_length);
}