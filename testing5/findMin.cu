#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMin.cuh"

__device__ static float atomicMin(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMin(float *dev_arr, int size, float *dev_min_val) {
	extern __shared__ float shmem[];
	const unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[i];

		__syncthreads();

		for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] > shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMin(dev_min_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMin(float *arr, int size, float *min_val) {
	float *dev_arr;
	float *dev_min_val;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_min_val, sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dev_min_val, 0.0, sizeof(float));
	
	findMin<<<1, size, size * sizeof(float)>>>(dev_arr, size, dev_min_val);
	
	hipMemcpy(min_val, dev_min_val, sizeof(float), hipMemcpyDeviceToHost);	
	hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_arr);
	hipFree(dev_min_val);

}
