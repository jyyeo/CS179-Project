#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void vecAdd (float *input1, float *input2, float *output, int size) {
	int tid = blockDim.x * gridDim.x + threadIdx.x;
	while (tid < size) {
		output[tid] = input1[tid] +input2[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cudaVecAdd (float *input1, float *input2, float *output, int size) {
	float* dev_input1;
	float* dev_input2;
	float* dec_input3;

	hipMalloc((void**)&dev_input1, size * sizeof(float));
	hipMalloc((void**)&dev_input2, size * sizeof(float));
	hipMalloc((void**)&dev_output, size * sizeof(float));

	hipMemcpy(dev_input1, input1, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_input2, input2, size * sizeof(float), hipMemcpyHostToDevice);

	vecAdd<<<1, size>>>(dev_input1, dev_input2, dev_output, size);

	hipMemcpy(output, dev_output, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_input1);
	hipFree(dev_input2);
	hipFree(dev_output);
}

int main(void) {
	float *input1, *input2, *output;
	int size = 10;

	input1 = (float*)malloc(size * sizeof(float));
	input2 = (float*)malloc(size * sizeof(float));

	for (int i = 0; i < size; i++) {
		input1[i] = i + 0.5;
		input2[i] = i + 1.5;
	}

	cudaVecAdd(input1, input2, output, size);

	for (int i = 0; i < size; i++) {
		printf("%f  %f\n", input1[i]+input2[i], output[i]);
	}
}