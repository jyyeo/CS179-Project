#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "vecAdd.cuh"

__global__ void vecAdd (float *input1, float *input2, float *output, int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	while (tid < size) {
		output[tid] = input1[tid] +input2[tid];
		tid += blockDim.x * gridDim.x;
	}
}

void cudaVecAdd (float *input1, float *input2, float *output, int size) {
	float* dev_input1;
	float* dev_input2;
	float* dev_output;

	hipMalloc((void**)&dev_input1, size * sizeof(float));
	hipMalloc((void**)&dev_input2, size * sizeof(float));
	hipMalloc((void**)&dev_output, size * sizeof(float));

	hipMemcpy(dev_input1, input1, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_input2, input2, size * sizeof(float), hipMemcpyHostToDevice);

	vecAdd<<<1, size>>>(dev_input1, dev_input2, dev_output, size);

	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_input1);
	hipFree(dev_input2);
	hipFree(dev_output);
}
