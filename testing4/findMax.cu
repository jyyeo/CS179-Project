#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMax(float *dev_arr, int size, float *dev_max_val) {
	extern __shared__ float shmem[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[tid];

		__syncthreads();

		for (int s = blockDim.x; s > 1; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] < shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMax(dev_max_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMax(float *dev_arr, int size, float *dev_max_val) {
	
	findMax<<<1, size>>>(dev_arr, size, dev_max_val);

	//std::cout << dev_output[0] << std::endl;

	hipFree(dev_arr);
	hipFree(dev_max_val);
}
