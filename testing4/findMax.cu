#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMax(float *dev_arr, int size, float *dev_max_val) {
	extern __shared__ float shmem[];
	const unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[i];

		__syncthreads();

		for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] < shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMax(dev_max_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMax(float *arr, int size, float *max_val) {
	float *dev_arr;
	float *dev_max_val;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_max_val, sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dev_max_val, 0.0, sizeof(float));
	findMax<<<1, size, size * sizeof(float)>>>(dev_arr, size, dev_max_val);
	
	hipMemcpy(max_val, dev_max_val, sizeof(float), hipMemcpyDeviceToHost);	
	hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_arr);
	hipFree(dev_max_val);

}
