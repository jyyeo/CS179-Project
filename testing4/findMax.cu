#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__global__ void findMax(float *dev_arr, int size, float *dev_output) {
	extern __shared__ float shmem[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		shmem[tid] = dev_arr[i];
	}

	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s && i < size) {
			shmem[tid] = fmaxf (shmem[tid],shmem[tid + s]);
		}
	}

	__syncthreads();

	if (tid == 0) {
		dev_output[blockIdx.x] = shmem[tid];
	}
}

void cudaFindMax(float *arr, int size, float *output) {
	float *dev_arr;
	float *dev_output;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_output, size * sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);

	findMax<<<1, size>>>(dev_arr, size, dev_output);

	//std::cout << dev_output[0] << std::endl;
	hipMemcpy(output, dev_output, size * sizeof(float), hipMemcpyDeviceToHost);	

	hipFree(dev_arr);
	hipFree(dev_output);
}
