#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMax(float *dev_arr, int size, float *dev_output) {
	extern __shared__ float shmem[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[i];

		__syncthreads();

		for (int s = blockDim.x; s > 1; s >>= 1) {
			if (tid < s && i < size) {
				if (shmem[tid] < shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMax(dev_output,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMax(float *arr, int size, float *output) {
	float *dev_arr;
	float *dev_output;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_output, 1 * sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);

	findMax<<<1, size>>>(dev_arr, size, dev_output);

	//std::cout << dev_output[0] << std::endl;
	hipMemcpy(output, dev_output, size * sizeof(float), hipMemcpyDeviceToHost);	

	hipFree(dev_arr);
	hipFree(dev_output);
}
