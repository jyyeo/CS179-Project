#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__device__ static float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void findMax(float *dev_arr, int size, float *dev_max_val) {
	extern __shared__ float shmem[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	while (i < size) {
		shmem[tid] = dev_arr[tid];

		__syncthreads();

		for (int s = blockDim.x; s > 1; s >>= 1) {
			if (tid < s) {
				if (shmem[tid] < shmem[tid + s]) {
					shmem[tid] = shmem[tid + s];
				}
			}
			__syncthreads();
		}

		if (tid == 0) {
			atomicMax(dev_max_val,shmem[0]);
		}
		i += blockDim.x * gridDim.x;
	}
}

void cudaFindMax(float *arr, int size, float *max_val) {
	float *dev_arr;
	float *dev_max_val;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_max_val, 1 * sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(dev_max_val, 0, sizeof(float));

	findMax<<<1, size>>>(dev_arr, size, dev_max_val);

	//std::cout << dev_output[0] << std::endl;
	hipMemcpy(&max_val, &dev_max_val, 1 * sizeof(float), hipMemcpyDeviceToHost);	

	hipFree(dev_arr);
	hipFree(dev_max_val);
}
