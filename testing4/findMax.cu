#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "findMax.cuh"

__global__ void findMax(float *dev_arr, int size, float *dev_output) {
	extern __shared__ float shmem[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		shmem[tid] = dev_arr[i];
	}

	__syncthreads();

	for (int j = 1; j < blockDim.x; j *= 2) {
		if (tid % (2*j) == 0) {
			shmem[tid] = fmaxf (shmem[tid],shmem[tid + j]);
		}
	}

	__syncthreads();

	if (tid == 0) {
		dev_output[blockIdx.x] = shmem[0];
	}
}

void cudaFindMax(float *arr, int size, float *output) {
	float *dev_arr;
	float *dev_output;

	hipMalloc((void**)&dev_arr, size * sizeof(float));
	hipMalloc((void**)&dev_output, size * sizeof(float));

	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);

	findMax<<<1, size>>>(dev_arr, size, dev_output);

	std::cout << dev_output[0] << std::endl;
	hipMemcpy(output, dev_output, size * sizeof(float), hipMemcpyDeviceToHost);	

	hipFree(dev_arr);
	hipFree(dev_output);
}