#include "addVec.cuh"
#include <hip/hip_runtime.h>

void cudaAddVecKernel(
	const float *input1,
	const float *input2,
	const float *output,
	const unsigned int size) 
{
	uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	while (thread_index < size) {
		output[thread_index] = input1[thread_index] + input2[thread_index];
		thread_index += blockDim.x * gridDim.x;
	}
}

void cudaCallAddVecKernel(const int blocks,
	const unsigned int threadsPerBlock,
	const float *input1,
	const float *input2,
	const float *output,
	const unsigned int size) 
{
	cudaAddVecKernel<<<blocks, threadsPerBlock>>>(input1, input2, size);
}
