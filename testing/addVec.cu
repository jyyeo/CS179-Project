#include <hip/hip_runtime.h>

__global__
void cudaAddVecKernel(
	const float *input1,
	const float *input2,
	float *output,
	const unsigned int size) 
{
	uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	while (thread_index < size) {
		output[thread_index] = input1[thread_index] + input2[thread_index];
		thread_index += blockDim.x * gridDim.x;
	}
}

void cudaCallAddVecKernel(const int blocks,
	const unsigned int threadsPerBlock,
	const float *input1,
	const float *input2,
	float *output,
	const unsigned int size) 
{
	cudaAddVecKernel<<<blocks, threadsPerBlock>>>(input1, input2, output, size);
}
